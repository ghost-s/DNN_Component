#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main() {
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);  // 获取设备数量

    if (deviceCount == 0) {
        std::cout << "No CUDA devices found!" << std::endl;
        return 0;
    }
    cout<<"Device Count: "<<deviceCount<<endl;

    int deviceId;
    hipGetDevice(&deviceId);  // 获取当前设备ID
    
    cout<<"Device ID: "<<deviceId<<endl;

    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceId);

    std::cout << "Max threads per block: " << maxThreadsPerBlock << std::endl;
    

    int maxThreadsDim[3];
    hipDeviceGetAttribute(&maxThreadsDim[0], hipDeviceAttributeMaxBlockDimX, deviceId);
    hipDeviceGetAttribute(&maxThreadsDim[1], hipDeviceAttributeMaxBlockDimY, deviceId);
    hipDeviceGetAttribute(&maxThreadsDim[2], hipDeviceAttributeMaxBlockDimZ, deviceId);

    std::cout << "Max block dimension (x, y, z): "
              << maxThreadsDim[0] << ", "
              << maxThreadsDim[1] << ", "
              << maxThreadsDim[2] << std::endl;

    

    int maxGridDim[3];
    hipDeviceGetAttribute(&maxGridDim[0], hipDeviceAttributeMaxGridDimX, deviceId);
    hipDeviceGetAttribute(&maxGridDim[1], hipDeviceAttributeMaxGridDimY, deviceId);
    hipDeviceGetAttribute(&maxGridDim[2], hipDeviceAttributeMaxGridDimZ, deviceId);

    std::cout << "Max grid dimension (x, y, z): "
              << maxGridDim[0] << ", "
              << maxGridDim[1] << ", "
              << maxGridDim[2] << std::endl;

    cout<<endl;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);  // 获取设备属性

    std::cout << "Device name: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max block dimensions (x, y, z): "
              << prop.maxThreadsDim[0] << ", "
              << prop.maxThreadsDim[1] << ", "
              << prop.maxThreadsDim[2] << std::endl;
    std::cout << "Max grid dimensions (x, y, z): "
              << prop.maxGridSize[0] << ", "
              << prop.maxGridSize[1] << ", "
              << prop.maxGridSize[2] << std::endl;

    

    return 0;
}
